#include "hip/hip_runtime.h"

/*
* Author: W. Evan Durno 
* Written April 2016 
* All rights reserved 
*/ 

#include <stdio.h> 
#include <float.h> 

////////////////////////////////////////////////////////////////////////////////////// external facing headers 

extern "C" { 

// GPU-accelerated multivariate regression with a factor model and floor student-t marginals via gaussian copula 
// all matrices are stores in column-major order 
// y : n X p matrix, counts data to be regressed 
// x : n X q matrix, design matrix   
// t : tn X K matrix, parameters, each column is a different parameter
// specifically, the order of parameters in one t column is beta_{q X p} , lsig_{p} , lnu_{p} , l_{p X m} , lpsi_{p} , always column-major order  
// m : the number of factors  
// p is assumed > 32, otherwise use of this software is not motivated 
// returns out : K-length vector of sample likelihoods 
void log_lik_cu ( float *y , float *x , float *t , int n , int p , int q , int tn , int K , int m , int n_iter , int seed , float *out ) ; 

} // end extern "C" 

////////////////////////////////////////////////////////////////////////////////////// local headers  

// stores device locations of major parameter components for the first parameter t 
// later elements can be accessed via column shifts 
struct params 
{ 
	float *beta ; 
	float *lsig ; 
	float *lnu ; 
	float *l ; 
	float *lpsi ; 
}; 

// Extracts parameters elements from a single parameter vector  
void construct_params ( float *t , struct params *a , int p , int q , int m ) ; 

// NOT USED   
// constructs I + L' inv(Psi) L 
// out : m X m X K matrix  
// threads : m X m X K  
__global__ void f2_make_inner_sig ( float *l , float *psi , float *out , int p , int m , int K , int tn ) ; 

// constructs F_M( y ) 
// mode : in { 1 , 2 , 3 } , indicating different parts of the argument domain  
// plus : an amount to add to y, either 0 or 1   
// out : n X p X K matrix   
// threads : n X p X K    
__global__ void f3_marginal_cdf ( float *y , float *x , float *beta , float *lsig , float *lnu , float *out , int n , int p , int q , int K , int tn , int mode , float plus ) ; 

// constructs diag( L L' + Psi )  
// out : p X K matrix  
// threads : p X K  
__global__ void f4_diag_sig ( float *l , float *psi , float *out , int p , int m , int K , int tn ) ; 

// NOT USED  
// constructs log det( L L' + Psi ) 
// out : K-length vector of matrix determinants  
__global__ void f5_ldet ( float *lpsi , float **ri , float *out , int m , int p , int K , int tn ) ; 

// NOT_USD  
// constructs inv( I + L' inv(Psi) L ) 
// in : product of f2 
// w : 4 X m X m X K matrix, working space 
// out : m X m X K matrix in out , but also m X m X K matrix in r 
// threads : K  
__global__ void f6_inv_inner_sig ( float *f2 , float *out , int m , int K , float *w , float **r ) ; 

// constructs log f_M(y) 
// in : f3( y ) , f3( y+1 ) 
// out : n X p X K matrix 
// threads : n X p X K  
__global__ void f7_lmarginal_pdf ( float *f3 , float *f31 , float *out , int n , int p , int K ) ; 

// NOT USED  
// constructs inv(Psi) - inv(Psi) L inv( I + L' inv(Psi) L ) L' inv(Psi) = inv(Sigma) 
// in : product of f6 
// out : p X p X K matrix   
// threads : p X p X K 
__global__ void f8_inv_sig ( float *f6 , float *l , float *lpsi , float *out , int p , int m , int K , int tn ) ; 

// constructs Sum_j log f_M( y_{ij} ) 
// in : product of f7 
// out : n X K matrix  
// threads n X K 
__global__ void f9_sum_lpdfs ( float *f7 , float *out , int n , int p , int K ) ; 

// constructs F_N^{-1}( F_M( y+1 ) ) 
// in : f3( y ) and product of f4 
// out : n X p X K matrix  
// threads : n X p X K  
__global__ void f10_F_N_inv ( float *f3 , float *f4 , float *out , int n , int p , int K ) ; 

// constructs log f_{N_p} ( f10 ) 
// out : n X K matrix  
// threads : n X K   
__global__ void f11_lpmnorm ( float *f10 , float *f101 , float *l , float *lpsi , float *out , int n , int p , int m , int K , size_t seed , int n_iter , float *w ) ; 

// constructs log_lik( t ; y , x ) 
// out : K-length vector   
// threads : K  
__global__ void f12_sum_log_likes ( float *f11 , float *f9 , float *out , int n , int K ) ;  

////////////////////////////////////////////////////////////////////////////////////// host implementations 

void log_lik_cu ( float *y , float *x , float *t , int n , int p , int q , int tn , int K , int m , int n_iter , int seed , float *out ) 
{ 
	size_t n_cores = 32 ;
        if( n_cores * 65536 < max( p*p*K , n*p*K ) ) 
                n_cores = max( p*p*K , n*p*K )/65536 + 1 ; 
        if( n_cores >= 1024 ) 
                n_cores = 1023 ; 
        if( n_cores * 65536 < max( p*p*K , n*p*K ) ) 
                fprintf( stderr , "CUDA WARNING: insufficient threads!\n" ) ; 
	
	hipError_t status = hipSuccess ; 
	
	// initialize streams 
	// s[0] : NOT USED  
	// s[1] : NOT USED  
	// s[2] : f7 - f9, f7 requires s[7:12]  
	// s[3] : f10(y), f10 requires s[7:9] as well as f4 via event[6]  
	// s[4] : f4 - f10(y+1), f10 requires s[10:12] 
	// s[5] : f11 - f12, f11 requires s[0], s[1], s[3], s[4], and f12 requires s[2]  
	// s[6] : NOT USED  
	// s[7] : f3(y,1) 
	// s[8] : f3(y,2) 
	// s[9] : f3(y,3) 
	// s[10] : f3(y+1,1) 
	// s[11] : f3(y+1,2) 
	// s[12] : f3(y+1,3) 
	hipStream_t stream[13] ; 
	int i ; 
	for( i = 0 ; i < 13 && status == hipSuccess ; i++ ) 
		status = hipStreamCreate( stream + i ) ; 
	
	// initialize events 
	// event[i] marks the completion of stream[i] 
	// except event[6] which marks the completion of f4 in s[4]  
	hipEvent_t event[13] ; 
	for( i = 0 ; i < 13 && status == hipSuccess ; i++ ) 
		status = hipEventCreate( event + i ) ; 
	
	// represent device memory 
	float *d_y = NULL ; 
	float *d_x = NULL ; 
	float *d_t = NULL ; 
	float *d_f3 = NULL ; 
	float *d_f31 = NULL ; 
	float *d_f4 = NULL ; 
	float **d_ri = NULL ; 
	float *d_f7 = NULL ; 
	float *d_f9 = NULL ; 
	float *d_f10 = NULL ; 
	float *d_f101 = NULL ; 
	float *d_f11 = NULL ; 
	float *d_f11w = NULL ; 
	float *d_f12 = NULL ; 
	
	// allocate device memory 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_y , n * p * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_x , n * q * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_t , tn * K * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_f3 , n * p * K * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_f31 , n * p * K * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_f4 , p * K * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_ri , K * sizeof(float*) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_f7 , n * p * K * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_f9 , n * K * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_f10 , n * p * K * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_f101 , n * p * K * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_f11 , n * K * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_f11w , m * n * K * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_f12 , K * sizeof(float) ) ; 
	
	// populate device memory 
	if( status == hipSuccess ) 
		status = hipMemcpy( d_y , y , n * p * sizeof(float) , hipMemcpyHostToDevice ) ; 
	if( status == hipSuccess ) 
		status = hipMemcpy( d_x , x , n * q * sizeof(float) , hipMemcpyHostToDevice ) ; 
	if( status == hipSuccess ) 
		status = hipMemcpy( d_t , t , tn * K * sizeof(float) , hipMemcpyHostToDevice ) ; 
	
	size_t free_mem , total_mem ; 
	if( status == hipSuccess ) 
		status = hipMemGetInfo( &free_mem, &total_mem ) ; 
	if( status == hipSuccess ) 
		fprintf( stderr , "Device free mem: %lu, total mem: %lu, remaining: %f%%\n" , free_mem , total_mem , ((float) free_mem)/((float) total_mem) ) ; 
	
	// identify parameter elements  
	struct params a ; 
	construct_params ( d_t , &a , p , q , m ) ; 
	
	// count threads 
	size_t t3 = n * p * K ; 
	size_t t4 = p * K ; 
	size_t t7 = n * p * K ; 
	size_t t9 = n * K ; 
	size_t t10 = n * p * K ; 
	size_t t11 = n * K ; 
	size_t t12 = K ;  
	
	///////////////// run and schedule kernels 
	
	// stream[7:9] 
	if( status == hipSuccess ) 
		f3_marginal_cdf <<< t3 / n_cores + 1 , n_cores , 0 , stream[7] >>> ( d_y , d_x , a.beta , a.lsig , a.lnu , d_f3 , n , p , q , K , tn , 1 , 0.0f ) ;  
	if( status == hipSuccess ) 
		f3_marginal_cdf <<< t3 / n_cores + 1 , n_cores , 0 , stream[8] >>> ( d_y , d_x , a.beta , a.lsig , a.lnu , d_f3 , n , p , q , K , tn , 2 , 0.0f ) ; 
	if( status == hipSuccess ) 
		f3_marginal_cdf <<< t3 / n_cores + 1 , n_cores , 0 , stream[9] >>> ( d_y , d_x , a.beta , a.lsig , a.lnu , d_f3 , n , p , q , K , tn , 3 , 0.0f ) ; 
	if( status == hipSuccess ) 
		status = hipEventRecord( event[7] , stream[7] ) ; 
	if( status == hipSuccess ) 
		status = hipEventRecord( event[8] , stream[8] ) ; 
	if( status == hipSuccess ) 
		status = hipEventRecord( event[9] , stream[9] ) ; 
	
	// stream[10:12] 
	if( status == hipSuccess ) 
		f3_marginal_cdf <<< t3 / n_cores + 1 , n_cores , 0 , stream[10] >>> ( d_y , d_x , a.beta , a.lsig , a.lnu , d_f31 , n , p , q , K , tn , 1 , 1.0f ) ; 
	if( status == hipSuccess ) 
		f3_marginal_cdf <<< t3 / n_cores + 1 , n_cores , 0 , stream[11] >>> ( d_y , d_x , a.beta , a.lsig , a.lnu , d_f31 , n , p , q , K , tn , 2 , 1.0f ) ; 
	if( status == hipSuccess ) 
		f3_marginal_cdf <<< t3 / n_cores + 1 , n_cores , 0 , stream[12] >>> ( d_y , d_x , a.beta , a.lsig , a.lnu , d_f31 , n , p , q , K , tn , 3 , 1.0f ) ; 
	if( status == hipSuccess ) 
		status = hipEventRecord( event[10] , stream[10] ) ; 
	if( status == hipSuccess ) 
		status = hipEventRecord( event[11] , stream[11] ) ; 
	if( status == hipSuccess ) 
		status = hipEventRecord( event[12] , stream[12] ) ; 
	
	// stream[2] 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[2] , event[7] , 0 ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[2] , event[8] , 0 ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[2] , event[9] , 0 ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[2] , event[10] , 0 ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[2] , event[11] , 0 ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[2] , event[12] , 0 ) ; 
	if( status == hipSuccess ) 
		f7_lmarginal_pdf <<< t7 / n_cores + 1 , n_cores , 0 , stream[2] >>> ( d_f3 , d_f31 , d_f7 , n , p , K ) ; 
	if( status == hipSuccess ) 
		f9_sum_lpdfs <<< t9 / n_cores + 1 , n_cores >>> ( d_f7 , d_f9 , n , p , K ) ; 
	if( status == hipSuccess ) 
		status = hipEventRecord( event[2] , stream[2] ) ; 
	
	// stream[4] 
	if( status == hipSuccess ) 
		f4_diag_sig <<< t4 / n_cores + 1 , n_cores , 0 , stream[4] >>> ( a.l , a.lpsi , d_f4 , p , m , K , tn ) ; 
	if( status == hipSuccess ) 
		status = hipEventRecord( event[6] , stream[4] ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[4] , event[10] , 0 ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[4] , event[11] , 0 ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[4] , event[12] , 0 ) ; 
	if( status == hipSuccess ) 
		f10_F_N_inv <<< t10 / n_cores + 1 , n_cores , 0 , stream[4] >>> ( d_f31 , d_f4 , d_f101 , n , p , K ) ; 
	if( status == hipSuccess ) 
		status = hipEventRecord( event[4] , stream[4] ) ; 
	
	// stream[3]  
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[3] , event[6] , 0 ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[3] , event[7] , 0 ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[3] , event[8] , 0 ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[3] , event[9] , 0 ) ; 
	if( status == hipSuccess ) 
		f10_F_N_inv <<< t10 / n_cores + 1 , n_cores , 0 , stream[3]  >>> ( d_f3 , d_f4 , d_f10 , n , p , K ) ; 
	if( status == hipSuccess ) 
		status = hipEventRecord( event[3] , stream[3] ) ; 
	
	// stream[5] 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[5] , event[3] , 0 ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[5] , event[4] , 0 ) ; 
	if( status == hipSuccess ) 
		f11_lpmnorm <<< t11 / n_cores + 1 , n_cores , 0 , stream[5] >>> ( d_f10 , d_f101 , a.l , a.lpsi , d_f11 , n , p , m , K , seed , n_iter , d_f11w ) ; 
	if( status == hipSuccess ) 
		status = hipStreamWaitEvent( stream[5] , event[2] , 0 ) ; 
	if( status == hipSuccess ) 
		f12_sum_log_likes <<< t12 / n_cores + 1 , n_cores >>> ( d_f11 , d_f9 , d_f12 , n , K ) ; 
	
	///////////////// finished running and scheduling kernels 
	
	// extract data from device 
	if( status == hipSuccess ) 
		status = hipMemcpy( out , d_f12 , K * sizeof(float) , hipMemcpyDeviceToHost ) ; 
	
	// delete streams 
	for( i = 0 ; i < 13 && status == hipSuccess ; i++ ) 
		status = hipStreamDestroy( stream[i] ) ; 
	
	// delete events 
	for( i = 0 ; i < 13 && status == hipSuccess ; i++ ) 
		status = hipEventDestroy( event[i] ) ; 
	
	// check for errors 
	if( status != hipSuccess ) 
		fprintf( stderr , "CUDA ERROR: %s\n" , hipGetErrorString(status) ) ;  
	
	// free device memory 
	hipFree( d_y ) ; 
	hipFree( d_x ) ; 
	hipFree( d_t ) ; 
	hipFree( d_f3 ) ; 
	hipFree( d_f31 ) ; 
	hipFree( d_f4 ) ; 
	hipFree( d_ri ) ; 
	hipFree( d_f7 ) ; 
	hipFree( d_f9 ) ; 
	hipFree( d_f10 ) ; 
	hipFree( d_f101 ) ; 
	hipFree( d_f11 ) ; 
	hipFree( d_f12 ) ; 
	hipFree( d_f11w ) ; 
	
	return ; 
} 

void construct_params ( float *t , struct params *a , int p , int q , int m ) 
{ 
	a->beta = t ; 
	a->lsig = t + p*q ; 
	a->lnu = t + p*(q + 1) ; 
	a->l = t + p*(q + 2) ; 
	a->lpsi = t + p*(q + 2 + m) ; 
} 

////////////////////////////////////////////////////////////////////////////////////// device implementations 

__global__ void f2_make_inner_sig ( float *l , float *psi , float *out , int p , int m , int K , int tn ) 
{ 
	int i = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( i >= m * m * K ) 
		return ; 
	int k = i / (m*m) ; 
	int col = (i - k*m*m) / m ; 
	int row = i - k*m*m - col*m ; 
	out += i ; 
	*out = 0.0f ; 
	for( i = 0 ; i < p ; i++ ) 
		*out += expf( -psi[ k*tn + i ]) * l[ k*tn + p*row + i ] * l[ k*tn + p*col + i ] ; 
	if( row == col ) 
		*out += 1.0f ; 
} 

__device__ double beta( float a , float b )
{
        return exp( lgammaf(a) + lgamma(b) - lgamma(a+b) ) ;
}

__device__ double beta_sr ( float x , float a , float b , float i )
{
        return expf( (a+i)*logf(x) + b*logf(1-x) - logf(a+i) + lgammaf(a+i+b) - lgammaf(a+i) - lgammaf(b) ) ;
}

__device__ void pt_beta_reductions ( float *x , float nu ) 
{
        int flag = ( (*x)*(*x) > nu )? 1 : 0 ;
        float y = ( flag )? nu/( (*x)*(*x) + nu) : 1.0f - 1.0f/( 1.0f + ((*x)/nu)*(*x) ) ;
        float a = ( flag )? 0.5f*nu : 0.5f ;
        float b = ( flag )? 0.5f : 0.5f*nu ;
        float out = 0.0f ;
        int i = 0 ;
        for( i = 0 ; i < 20 ; i++ ) 
        { 
                out += beta_sr( y , a , b , (float) i ) ;
        } 
        out = ( flag )? 1.0f - 0.5f*out : 1.0f - 0.5f*(1.0f - out) ;
        out = ( *x < 0.0 )? 1.0f - out : out ;
        *x = out ;
}

__device__ void pt_alg395 ( float *x , float nu )
{
        float t = (*x) * (*x) ;
        float y = t/nu ;
        float b = 1.0f + y ;
        y = ( y > 1e-6f )? logf(b) : y ;
        float a = nu - 0.5 ;
        b = 48.0f * a * a ;
        y = a * y ;
        y = (((((-0.4f*y-3.3f)*y - 24.0f)*y - 85.5f)/(0.8f*y*y + 100.0f + b) + y + 3.0f)/b + 1.0f)*sqrtf(y) ;
        y = normcdff( y ) ;
        *x = ( *x > 0.0f )? y : 1.0f - y ;
}

__device__ void pt_normal ( float *x ) 
{
        *x = normcdff( *x ) ;
}

__global__ void f3_marginal_cdf ( float *y , float *x , float *beta , float *lsig , float *lnu , float *out , int n , int p , int q , int K , int tn , int mode , float plus ) 
{ 
	int i = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( i >= n * p * K ) 
		return ; 
	int k = i / (p*n) ; 
	int col = (i - k*p*n) / n ; // dimension  
	int row = i - k*p*n - col*n ; // sample  
	if( mode == 1 ) 
	{ 
		if( expf( lnu[ tn*k + col ] ) <= 1e5f ) // must satisfy to proceed: nu > 1e5f 
			return ; 
		y += n*col + row ; 
		out += i ; 
		if( *y + plus <= 0.0f ) // if y is zero, all forms of f3 return 0.0f. We do this under the 'mode ==' test to avoid write collisions. 
		{ 
			*out = 0.0f ; 
			return ; 
		} 
		float sig = expf( lsig[ tn*k + col ] ) ; 
		float mu = 0.0f ; 
		for( i = 0 ; i < q ; i++ ) 
			mu += x[ i*n + row ] * beta[ k*tn + col*q + i ] ; 
		*out = logf((*y) + plus)/sig - mu ; 
		pt_normal( out ) ;  
	} 
	if( mode == 2 ) 
	{ 
                float nu = expf( lnu[ tn*k + col ] ) ;
                if( nu > 1e5f ) // must satisfy to proceed: nu <= 1e5f  
                        return ;
                y += n*col + row ;
                float sig = expf( lsig[ tn*k + col ] ) ;
                float mu = 0.0f ; 
		int j ; 
                for( j = 0 ; j < q ; j++ )
                        mu += x[ j*n + row ] * beta[ k*tn + col*q + j ] ;
                float tmp = logf((*y) + plus)/sig - mu ;
                if( ! ( nu > 200.0f || ( tmp*tmp < nu && nu > 4.0f ) ) ) 
                        return ;
                out += i ;
                if( *y + plus <= 0.0f ) 
                {
                        *out = 0.0f ;
                        return ;
                }
                *out = tmp ;
                pt_alg395 ( out , nu ) ;
	} 
	if( mode == 3 ) 
	{ 
                float nu = expf( lnu[ tn*k + col ] ) ;
                if( nu > 1e5f )  
                        return ;
                y += n*col + row ;
                float sig = expf( lsig[ tn*k + col ] ) ;
                float mu = 0.0f ; 
		int j ; 
                for( j = 0 ; j < q ; j++ )
                        mu += x[ j*n + row ] * beta[ k*tn + col*q + j ] ;
                float tmp = logf((*y) + plus)/sig - mu ;
                if( nu > 200.0f || ( tmp*tmp < nu && nu > 4.0f ) )       
                        return ;
                out += i ;
                if( *y + plus <= 0.0f ) 
                {
                        *out = 0.0f ;
                        return ;
                }
                *out = tmp ;
                pt_beta_reductions ( out , nu ) ; 
	} 
} 

__global__ void f4_diag_sig ( float *l , float *psi , float *out , int p , int m , int K , int tn ) 
{ 
	int i = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( i >= p * K ) 
		return ; 
	int k = i / p ; 
	int row = i - k*p ; // dimension  
	out += i ; // row-th row in the k-th vector  
	*out = expf( psi[ k*tn + row ] ) ; 
	for( i = 0 ; i < m ; i++ ) 
		*out += l[ k*tn + i*p + row ] * l[ k*tn + i*p + row ] ;  
} 

__global__ void f5_ldet ( float *lpsi , float **ri , float *out , int m , int p , int K , int tn ) 
{ 
	int i = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( i >= K ) 
		return ; 
	int k = i ; 
	float *invR = ri[i] ; 
	out += i ; 
	*out = 0.0f ; 
	for( i = 0 ; i < p ; i++ ) 
		*out += lpsi[ k*tn + i ] ; // returns on log scale  
	for( i = 0 ; i < m ; i++ ) 
		*out -= logf( fabs(invR[ i*m + i ]) ) ; // algebraically correct and numerically stable (avoids squaring)   
} 

__device__ void copyFloats ( float *dest , float *src , int n )
{
        float *fin = dest + n ;
        while( dest < fin )
        {
                *dest = *src ;
                dest++ ;
                src++ ;
        }
}

// single-threaded matrix product 
// utilizes a single cuda-core, meant for mass products 
// x : nXn matrix 
// y : nXn matrix 
// w : nXn matrix, working space 
// returns in w 
__device__ void mat_prod_serial ( float *x , float *y , int n , float *w )
{
        float *fin = x + n*n ;
        float *t1 = NULL ;
        float *t2 = NULL ;
        int i , j ;
        for( j = 0 ; j < n ; j++ )
        {
                for( i = 0 ; i < n ; i++ )
                {
                        t1 = x + i ; // i-th row of x  
                        t2 = y + j*n ; // j-th column of y 
                        *w = 0.0f ;
                        while( t1 < fin )
                        {
                                *w += (*t1) * (*t2) ;
                                t1 += n ;
                                t2 ++ ;
                        }
                        w++ ;
                }
        }
}

// Householder reflection matrix 
// Creates the (nx-nv+1)-th Householder reflection matrix for QR decomposition 
// x : output matrix, nx X nx 
// v : vector of length nv  
// requires nv <= nx  
__device__ void construct_reflection_matrix ( float *x , float *v , int nx , int nv )
{
        int N = nx - nv ;
        int i , j ;
        for( i = 0 ; i < nx ; i++ )
        {
                for( j = 0 ; j < nx ; j++ )
                {
                        if( i == j )
                                x[ i + nx*i ] = 1.0f ;
                        else
                                x[ i + nx*j ] = 0.0f ;
                        if( i >= N && j >= N ) // less 2 X v's outer product  
                        {
                                x[ i + nx*j ] -= 2.0f * v[ i-N ] * v[ j-N ] ;
                        }
                }
        }
}

// invert upper-triangular r_{n X n} into ri with back fitting  
__device__ void invert_upper_serial ( float *r , int n , float *ri )
{
        int i , j ; // row and column of ri respectively 
        int k ;
        for( j = 0 ; j < n ; j++ )
        {
                for( i = n-1 ; i >= 0 ; i-- )
                {
                        if( i > j ) // lower quadrant 
                                ri[ i + j*n ] = 0.0f ;
                        else if( i == j )
                                ri[ i + j*n ] = 1.0f / r[ i + j*n ] ;
                        else // i < j 
                        {
                                ri[ i + j*n ] = 0.0f ;
                                for( k = j ; k > i ; k-- )
                                        ri[ i + j*n ] -= r[ i + k*n ] * ri[ k + j*n ] ;
                                ri[ i + j*n ] /= r[ i + i*n ] ;
                        }
                }
        }
}

__global__ void f6_inv_inner_sig ( float *f2 , float *out , int m , int K , float *w , float **ri ) 
{ 
	int i = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( i >= K ) 
		return ; 
	out += i*m*m ; 
	if( m == 1 ) 
	{ 
		*out = 1.0f / (*out) ; 
		return ; 
	} 
        int work_n = 4*m*m ; // total working space size per thread  
        float *qt = w + i * work_n ; // transpose of Q matrix; product of QR decomposition 
        float *qi = w + i*work_n + m*m ; // Q_i workingspace for QR decomp 
        ri[i] = w + i*work_n + 2*m*m ; // space for R^{-1} after QR decomp 
        float *w2 = w + i*work_n + 3*m*m ; // secondary working space 
        float *v = w2 ; // different name for same space, stands in for a vector. Optimizer should remove at compile 
        float mag = -1.0f ; // magnitude 
	
	// copy f1 to out 
	f2 += i*m*m ; 
	copyFloats ( out , f2 , m*m ) ; 
	
        // set qt to identity  
        int ii , j ;
        for( ii = 0 ; ii < m ; ii++ )
        { 
                for( j = 0 ; j < m ; j++ )
                {
                        if( ii == j )
                                qt[ ii + j*m ] = 1.0f ;
                        else
                                qt[ ii + j*m ] = 0.0f ;
                }
        } 
	
	// QR decomposition via Householder reflections 
        for( ii = 0 ; ii < m ; ii++ ) 
        {
                // calc rotation vector  
                for( j = 0 ; j < m - ii ; j++ )
                        v[j] = out[ ii + j + ii*m ] ;
                mag = 0.0f ;
                for( j = 0 ; j < m - ii ; j++ )
                        mag += v[j]*v[j] ;
                mag = sqrtf(mag) ;
                v[0] += copysignf( mag , v[0] ) ;
                mag = 0.0f ;
                for( j = 0 ; j < m - ii ; j++ )
                        mag += v[j]*v[j] ;
                mag = sqrtf( mag ) ;
                for( j = 0 ; j < m - ii ; j++ )
                        v[j] /= mag ;

                construct_reflection_matrix( qi , v , m , m-ii ) ;

                // update m = Qi m 
                mat_prod_serial ( qi , out , m , w2 ) ; // optional todo : force zeros below i-th entry of i-th column 
                copyFloats ( out , w2 , m*m ) ; // m stores r 

                // update qt = qi qt 
                mat_prod_serial ( qi , qt , m , w2 ) ;
                copyFloats ( qt , w2 , m*m ) ;
        }

        // Backfit R to I, producing a numerically stable inverse 
        invert_upper_serial ( out , m , ri[i] ) ;

        // Matrix product R^{-1} Q^T produces the inverse of m  
        mat_prod_serial ( ri[i] , qt , m , out ) ;
} 

__global__ void f7_lmarginal_pdf ( float *f3 , float *f31 , float *out , int n , int p , int K ) 
{ 
	int i = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( i >= n*p*K ) 
		return ; 
	int k = i / (n*p) ; 
	int col = (i - k*n*p) / n ; 
	int row = i - k*n*p - col*n ; 
	out += i ; // k-th matrix, row-th row, col-th col 
	*out = fabsf( f31[ k*n*p + col*n + row ] - f3[ k*n*p + col*n + row ] ) ; // I expect round-offs in the tail 
	*out = ( *out <= 0.0f ) ? sqrtf(FLT_MIN) : *out ; // just in case 
	*out = logf(*out) ;  
} 

__global__ void f8_inv_sig ( float *f6 , float *l , float *lpsi , float *out , int p , int m , int K , int tn ) 
{ 
	int i = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( i >= p*p*K ) 
		return ; 
	int k = i / (p*p) ; 
	int col = (i - k*p*p) / p ; 
	int row = i - k*p*p - col*p ; 
	out += i ; // k-th matrix, col-th col, row-th row 
	int j ; 
	*out = 0.0f ; 
	for( i = 0 ; i < m ; i++ ) 
	{ 
		for( j = 0 ; j < m ; j++ ) 
			*out += l[ k*tn + j*p + row ] * f6[ k*m*m + j*m + i ] * l[ k*tn + i*p + col ] ; 
	} 
	*out *= expf( -lpsi[k*tn + row] - lpsi[k*tn + col] ) ; 
	*out = ( row == col ) ? expf( -lpsi[ k*tn + row ] ) - (*out) : -(*out) ; 
} 

__global__ void f9_sum_lpdfs ( float *f7 , float *out , int n , int p , int K ) 
{ 
	int i = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( i >= n*K ) 
		return ; 
	int k = i / n ; 
	int row = i - k*n ; 
	out += i ; // k-th vector, row-th row  
	*out = 0.0f ; 
	for( i = 0 ; i < p ; i++ ) 
		*out += f7[ k*n*p + i*n + row ] ;  
} 

__global__ void f10_F_N_inv ( float *f3 , float *f4 , float *out , int n , int p , int K ) 
{ 
	int i = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( i >= n * p * K ) 
		return ; 
	int k = i / (n*p) ; 
	int col = (i - k*n*p) / n ; 
	int row = i - k*n*p - col*n ; 
	out += i ; // k-th matrix, col-th col, row-th row 
	if( f3[ k*n*p + col*n + row ] <= 0.0f ) 
		*out = -FLT_MAX ; 
	else if( f3[ k*n*p + col*n + row ] >= 1.0f ) 
		*out = FLT_MAX ; 
	else 
		*out = sqrtf( f4[ k*p + col ] ) * normcdfinvf( f3[ k*n*p + col*n + row ] ) ;  
} 

// Add log scale values and return on log scale 
// Used to avoid under/over-flows   
// returns log( a + b )  
__device__ float logSum ( float a , float b ) 
{ 
	if( a < b ) // exponents can't be allowed to get too large  
	{ 
		float t = a ; 
		a = b ; 
		b = t ; 
	} 
	return a + logf( 1.0f + expf( b - a ) ) ;  
} 

// Subtract log scale values from one another 
// returns log( a - b )   
__device__ float logSubtract( float a , float b ) 
{ 
	return a + logf( 1.0f - expf( b - a ) ) ; 
} 

// a mod b  
__device__ size_t mod( size_t a , size_t b ) 
{ 
	return a - b*(a/b) ; 
} 

__device__ float generate_unif_lcg( size_t *seed ) 
{ 
	*seed = mod( 1103515245*(*seed) + 12345 , 2147483648 ) ; 
	return ((float) ((*seed)) + 10000) / 2147503648.0f ; // = 2147483648 + 20000 ; do not return 0 or 1 
} 

__device__ float truncate ( float x ) 
{ 
	return copysignf( floorf( fabsf(x) ) , x ) ; 
} 

// Returns log( normal_cdf(x) )  
__device__ float log_norm_cdf ( float x ) 
{ 
	// Catch boundary cases  
	if( x >= 1e37 ) 
		return 0.0f ;  
	if( x <= -1e37 ) 
		return -1.0f/0.0f ; 
	// Use pre-written CUDA software when possible  
	if( x <= 5.657f && x >= -5.657f ) // sqrt(32)  
		return logf(normcdff(x)) ; 
	// else : I borrowed this from R's prorm.c, references were ungiven for algorithm content  
	float p[6] = {
        0.21589853405795699f ,
        0.1274011611602473639f ,
        0.022235277870649807f ,
        0.001421619193227893466f ,
        2.9112874951168792e-5f ,
        0.02307344176494017303f }; 
	float q[5] = { 
        1.28426009614491121f ,
        0.468238212480865118f ,
        0.0659881378689285515f ,
        0.00378239633202758244f ,
        7.29751555083966205e-5f }; 
	float xsq = 1/(x*x) ; 
	float xnum = p[5]*xsq ; 
	float xden = xsq ; 
	int i ; 
	for( i = 0 ; i < 4 ; i++ ) 
	{ 
		xnum = ( xnum + p[i] ) * xsq ; 
		xden = ( xden + q[i] ) * xsq ; 
	} 
	float tmp = xsq * ( xnum + p[4] ) / ( xden + q[4] ) ; 
	tmp = ( 0.3989422804014327 - tmp ) / fabsf(x) ; 
	
//	xsq = truncate( x * 16.0f ) / 16.0f ; // modified from original code   
	xsq = truncate( x * 64.0f ) / 64.0f ; 
	float del = (x - xsq) * (x + xsq) ; 
	
	if( x < 0.0f ) 
		return ( -xsq * xsq * 0.5f ) + ( -del * 0.5f ) + logf(tmp) ; 
	return logf( 1.0f - expf( -xsq * xsq * 0.5f )*expf( -del * 0.5f ) * tmp ) ; 
} 

__global__ void f11_lpmnorm ( float *f10 , float *f101 , float *l , float *lpsi , float *out , int n , int p , int m , int K , size_t seed , int n_iter , float *w ) 
{ 
	int i = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( i >= n * K ) 
		return ; 
	int k = i / n ; 
	int row = i - k*n ; 
	out += i ; // k-th vector, row-th row 
	w += m*i ; 
	size_t local_seed = seed + i ; 
	float lo, hi, mu , tmp , swp ; 
	float sqMin = sqrt(FLT_MIN) ; 
	*out = FLT_MIN ; 
	int j , kk ; 
	for( i = 0 ; i < n_iter ; i++ ) // MC-integral, E[ P( N in [a,b] | F ) ] = P( N in [a,b] ) 
	{ 
		for( kk = 0 ; kk < m ; kk++ ) 
			w[kk] = normcdfinvf( generate_unif_lcg( &local_seed ) ) ; // generate a standard normal 
		tmp = 0.0f ; 
		for( j = 0 ; j < p ; j++ ) 
		{ 
			mu = 0.0f ; 
			for( kk = 0 ; kk < m ; kk++ ) 
				mu += l[ k*p*m + kk*p + j ] * w[kk] ; 
			lo = (f10[ k*n*p + j*n + row ] - mu) * fmaxf( sqMin , expf( -lpsi[ k*p + j ] ) ); 
			hi = (f101[ k*n*p + j*n + row ] - mu) * fmaxf( sqMin , expf( -lpsi[ k*p + j ] ) ) ; 
			if( hi < lo ) // possible via roundoffs  
			{ 
				swp = lo ; // use as temp variable  
				lo = hi ; 
				hi = swp ; 
			} 
			if( 0.0f < lo ) // both positive likely results in an underflow 
			{ 
				swp = lo ; 
				lo = -hi ; 
				hi = -swp ; 
			} 
			swp = logSubtract( log_norm_cdf(hi) , log_norm_cdf(lo) ) ; 
			if( swp != swp ) // round-off ! 
				tmp += -FLT_MAX/1000.0f ; 
			else 
				tmp += swp ;  
		} 
		if( i == 1 ) 
			*out = tmp ; 
		else  
			*out = logSum( *out , tmp ) ; 
	} 
	*out -= logf( (float) n_iter ) ; 
} 

__global__ void f12_sum_log_likes ( float *f11 , float *f9 , float *out , int n , int K ) 
{ 
	int i = blockIdx.x * blockDim.x + threadIdx.x ; 
	if( i >= K ) 
		return ; 
	out += i ; // i-th entry 
	*out = 0.0f ; 
	int j ; 
	for( j = 0 ; j < n ; j++ ) 
	{ 
		*out += f11[ i*n + j ] + f9[ i*n + j ] ; 
	} 
}  














