

#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

void loadData( char* fileName , int n , float* x , float* y , float* mass , int* actual ) ;
float getVal( char* str , int start , int subLen ) ; 
__global__ void iter( int n , float* xVel , float* yVel ,  float* x , float* y , float* mass , float G , float delt , float* xOut , float* yOut ) ; 

int main( int argc , char** argv ) 
{
	if( argc < 4 ) 
	{
		printf( "REQUIRED: Please provide (arg1) a file to load, (arg2) number of bodies, and (arg3) the number of iterations\n" ) ; 
		printf( "OPTIONAL: (arg4) time-step size in seconds (default=1000.0)\n" ) ; 
		return( 0 ) ; 
	}
	
	float G = 6.67384 * ((float) pow( 10.0 , -11.0 )) ; // Newton's gravitational constant 
	
	int maxIter = atoi( argv[3] ) ; 
	float delt = 1000.0 ; 
	if( argc > 4 ) 
		delt = atof( argv[4] ) ; 
	
	int n = atoi( argv[2] ) ; 
	int m ; 
	float* x = (float*) malloc( n * sizeof(float) ) ; 
	float* y = (float*) malloc( n * sizeof(float) ) ; 
	float* mass = (float*) malloc( n * sizeof(float) ) ; 
	loadData( argv[1] , n , x , y , mass , &m ) ; 
	
	struct timeval startALL, endALL ;
	gettimeofday( &startALL , NULL ) ; 
	
	////////////////// ALLOCATE MEMORY ON DEVICE PRIOR to COMPUTATION 
	float* d_x ; 
	float* d_y ; 
	float* d_mass ; 
	float* d_xVel ; 
	float* d_yVel ; 
	float* d_xTemp ; 
	float* d_yTemp ; 
	float* zeros = (float*) malloc( n * sizeof(float) ) ; 
	int i ; 
	for( i = 0 ; i < n ; i ++ ) 
		zeros[i] = 0.0 ; 
	
	hipError_t status = hipMalloc( &d_x , n * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_y , n * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_mass , n * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_xVel , n * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_yVel , n * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_xTemp , n * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMalloc( &d_yTemp , n * sizeof(float) ) ; 
	if( status == hipSuccess ) 
		status = hipMemcpy( d_x , x , n * sizeof(float) , hipMemcpyHostToDevice ) ; 
	if( status == hipSuccess ) 
		status = hipMemcpy( d_y , y , n * sizeof(float) , hipMemcpyHostToDevice ) ; 
	if( status == hipSuccess ) 
		status = hipMemcpy( d_mass , mass , n * sizeof(float) , hipMemcpyHostToDevice ) ; 
	if( status == hipSuccess ) 
		status = hipMemcpy( d_xVel , zeros , n * sizeof(float) , hipMemcpyHostToDevice ) ; 
	if( status == hipSuccess ) 
		status = hipMemcpy( d_yVel , zeros , n * sizeof(float) , hipMemcpyHostToDevice ) ; 
	
	//////////////// ASYNCHRONOUS QUEUE JOBS ON DEVICE 
	
	struct timeval startKERNEL , endKERNEL ; 
	gettimeofday( &startKERNEL , NULL ) ; 
	
	for( i = 0 ; i < maxIter ; i+=2 ) 
	{
		iter<<< m/32+1 , 32 >>>( m , d_xVel , d_yVel , d_x , d_y , d_mass , G , delt , d_xTemp , d_yTemp ) ; 
		iter<<< m/32+1 , 32 >>>( m , d_xVel , d_yVel , d_xTemp , d_yTemp , d_mass , G , delt , d_x , d_y ) ; 
	}
	
	if( status == hipSuccess ) ;
                 status = hipDeviceSynchronize() ;
	
	gettimeofday( &endKERNEL , NULL ) ; 
	fprintf( stderr , "GPU kernel time: %ld microseconds\n", ((endKERNEL.tv_sec * 1000000 + endKERNEL.tv_usec)
                  - (startKERNEL.tv_sec * 1000000 + startKERNEL.tv_usec)));
	
	//////////////// COPY MEMORY TO HOST
	
	if( status == hipSuccess ) 
		status = hipMemcpy( x , d_x , n * sizeof(float) , hipMemcpyDeviceToHost ) ; 
	if( status == hipSuccess ) 
		status = hipMemcpy( y , d_y , n * sizeof(float) , hipMemcpyDeviceToHost ) ; 
	
	if( status != hipSuccess )
                printf( "ERROR: %s\n" , hipGetErrorString(status) ) ; 
	
	hipFree( d_x ) ; 
	hipFree( d_y ) ; 
	hipFree( d_mass ) ; 
	hipFree( d_xVel ) ; 
	hipFree( d_yVel ) ; 
	hipFree( d_xTemp ) ; 
	hipFree( d_yTemp ) ; 
	
	gettimeofday( &endALL , NULL ) ; 
	fprintf( stderr , "GPU kernel and comm time: %ld microseconds\n", ((endALL.tv_sec * 1000000 + endALL.tv_usec)
                  - (startALL.tv_sec * 1000000 + startALL.tv_usec)));
	
	hipDeviceReset() ; 
	
	for( i = 0 ; i < m ; i++ ) 
		printf( "%f\t%f\n" , x[i] , y[i] ) ; 
	
	return( 0 ) ; 
}

__global__ void iter( int n , float* xVel , float* yVel ,  float* x , float* y , float* mass , float G , float delt , float* xOut , float* yOut )
{
	int rank = threadIdx.x + blockIdx.x * blockDim.x ; 
	if( rank >= n ) 
		return ; 
        int i ;
        float r ;
        float xForce = 0.0 ;
        float yForce = 0.0 ;
        for( i = 0 ; i < n ; i++ )
        {
                if( i != rank )
                {
                        // Calculations are done in exponentiated logs to reduce roundoffs
                        r = sqrt( (x[i] - x[rank])*(x[i] - x[rank]) + (y[i] - y[rank])*(y[i] - y[rank]) ) ;
                        if( x[i] > x[rank] )
                                xForce = xForce + exp( log(G) + log(mass[i]) + log(mass[rank]) + log( x[i] - x[rank] ) - 3.0*log( r ) ) ;
                        if( x[i] < x[rank] )
                                xForce = xForce - exp( log(G) + log(mass[i]) + log(mass[rank]) + log( x[rank] - x[i] ) - 3.0*log( r ) ) ;
                        // case: x[i] == x[rank] : do nothing
                        if( y[i] > y[rank] )
                                yForce = yForce + exp( log(G) + log(mass[i]) + log(mass[rank]) + log( y[i] - y[rank] ) - 3.0*log( r ) ) ;
                        if( y[i] < y[rank] )
                                yForce = yForce - exp( log(G) + log(mass[i]) + log(mass[rank]) + log( y[rank] - y[i] ) - 3.0*log( r ) ) ;
                        // case: y[i] == y[rank] : do nothing
                }
        }
        xVel[rank] = xVel[rank] + xForce * delt / mass[rank] ;
        yVel[rank] = yVel[rank] + yForce * delt / mass[rank] ;
        xOut[rank] = x[rank] + xVel[rank] * delt ;
        yOut[rank] = y[rank] + yVel[rank] * delt ;
}

void loadData( char* fileName , int n , float* x , float* y , float* mass , int* actual )
{
        char temp[1000] ;
        FILE *file ;
        file = fopen( fileName , "r" ) ;
        if( file == NULL )
        {
                printf( "File failed to open!\n" ) ;
                return ;
        }

        *actual = 0 ;
        int delim1 , delim2 , delim3 ; // ends of delimeters
        int len , i , j , flag ;
        // char temp1[1000] ;
        for( j = 0 ; fgets( temp , 1000 , file ) != NULL && j < n ; j++ )
        {
                len = strlen( temp ) ;
                delim1 = -1 ;
                for( i = 0 ; i < len && delim1 < 0 ; i++ )
                {
                        if( temp[i] != ' ' )
                                delim1 = i ;
                }
                delim2 = -1 ;
                flag = -1 ;
                for( i = delim1 + 1 ; i < len && delim2 < 0 ; i++ )
                {
                        if( temp[i] == ' ' )
                                flag = 1 ;
                        if( temp[i] != ' ' && flag > 0 )
                                delim2 = i ;
                }
                delim3 = -1 ;
                flag = -1 ;
                for( i = delim2 + 1 ; i < len && delim3 < 0 ; i++ )
                {
                        if( temp[i] == ' ' )
                                flag = 1 ;
                        if( temp[i] != ' ' && flag > 0 )
                                delim3 = i ;
                }
                if( delim1 < 0 || delim2 < 0 || delim3 < 0 )
                {
                        printf( "Input data formatting error\n" ) ;
                        return ;
                }

                x[j] = getVal( temp , delim1 , delim2 - delim1 ) ;
                y[j] = getVal( temp , delim2 , delim3 - delim1 ) ;
                mass[j] = getVal( temp , delim3 , -1 ) ;
		
		*actual = *actual + 1 ;
        }
        fclose( file ) ;
}

float getVal( char* str , int start , int subLen )
{
        int len = strlen( str ) ;
        if( subLen < 0 )
                subLen = len - start + 1 ;
        else
                subLen = subLen + 1 ;
        char temp[subLen] ;
        temp[subLen - 1] = '\0' ;
        int i ;
        for( i = 0 ; i < subLen - 1 ; i++ )
        {
                temp[i] = str[i+start] ;
        }
        return( atof( temp ) ) ;
}

